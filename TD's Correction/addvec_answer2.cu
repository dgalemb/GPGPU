
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vector_add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

/* experiment with N */
/* how large can it be? */
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	hipEventRecord(start, 0);
	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to copy %3.1f ms\n", elapsedTime);

	hipEventRecord(start, 0);
	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	vector_add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to execute %3.1f ms\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf( "c[0] = %d\n",c[0] );
	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
