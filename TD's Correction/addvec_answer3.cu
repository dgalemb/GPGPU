
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CREATE_CUDAEVENT hipEvent_t start, stop; hipEventCreateate(&start); hipEventCreateate(&stop);
#define START_CUDAEVENT hipEventRecord(start, 0);
#define STOP_AND_PRINT_CUDAEVENT(txt) hipEventRecord(stop, 0);\
cudaEventSynchronize(stop);\
{float elapsedTime;\
cudaEventElapsedTime(&elapsedTime, start, stop);\
printf("Time to %s %3.1f ms\n", #txt, elapsedTime);}


__global__
void vector_add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

__global__
void vector_add_UM(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

/* experiment with N */
/* how large can it be? */
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
	CREATE_CUDAEVENT;
	int size = N * sizeof( int );

	/*@ CpMem Classical Section @*/
	printf(">>> Results for MemCopy\n");
	int *h_a, *h_b, *h_c;
	int *d_a, *d_b, *d_c;

	START_CUDAEVENT
	h_a = (int *)malloc( size );
	h_b = (int *)malloc( size );
	h_c = (int *)malloc( size );
	STOP_AND_PRINT_CUDAEVENT([Classical] host allocation)

	START_CUDAEVENT
	for( int i = 0; i < N; i++ )
	{
		h_a[i] = h_b[i] = i;
		h_c[i] = 0;
	}
	STOP_AND_PRINT_CUDAEVENT([Classical] Initialize)

	START_CUDAEVENT
	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );
	STOP_AND_PRINT_CUDAEVENT([Classical] device allocation)

	START_CUDAEVENT
	hipMemcpy( d_a, h_a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, h_b, size, hipMemcpyHostToDevice );	
	STOP_AND_PRINT_CUDAEVENT([Classical] MemCopy Host to Device)

	START_CUDAEVENT
	vector_add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );
	STOP_AND_PRINT_CUDAEVENT([Classical] execution)

	START_CUDAEVENT
	hipMemcpy( h_c, d_c, size, hipMemcpyDeviceToHost );
	STOP_AND_PRINT_CUDAEVENT([Classical] MemCopy Device to Host)

	free(h_a);
	free(h_b);
	free(h_c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	/*@ End CpMem Classical Section @*/

	/*@ CpMem Pinned Section @*/
	printf("\n>>> Results for Pinned Memory\n");
	int *h_ap, *h_bp, *h_cp;
	int *d_ap, *d_bp, *d_cp;

	START_CUDAEVENT
	hipHostAlloc((void **) &h_ap, size, hipHostMallocDefault);
	hipHostAlloc((void **) &h_bp, size, hipHostMallocDefault);
	hipHostAlloc((void **) &h_cp, size, hipHostMallocDefault);
	STOP_AND_PRINT_CUDAEVENT([Pinned] host allocation)
	
	START_CUDAEVENT
	for( int i = 0; i < N; i++ )
	{
		h_ap[i] = h_bp[i] = i;
		h_cp[i] = 0;
	}
	STOP_AND_PRINT_CUDAEVENT([Pinned] initialize)

	START_CUDAEVENT
	hipMalloc( (void **) &d_ap, size );
	hipMalloc( (void **) &d_bp, size );
	hipMalloc( (void **) &d_cp, size );
	STOP_AND_PRINT_CUDAEVENT([Pinned] device allocation)

	START_CUDAEVENT
	hipMemcpy( d_ap, h_ap, size, hipMemcpyHostToDevice );
	hipMemcpy( d_bp, h_bp, size, hipMemcpyHostToDevice );	
	STOP_AND_PRINT_CUDAEVENT([Pinned] MemCopy Host to Device)

	START_CUDAEVENT
	vector_add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_ap, d_bp, d_cp );
	STOP_AND_PRINT_CUDAEVENT([Pinned] execution)

	START_CUDAEVENT
	hipMemcpy( h_cp, d_cp, size, hipMemcpyDeviceToHost );
	STOP_AND_PRINT_CUDAEVENT([Pinned] MemCopy Device to Host)

	hipHostFree(h_ap);
	hipHostFree(h_bp);
	hipHostFree(h_cp);
	hipFree( d_ap );
	hipFree( d_bp );
	hipFree( d_cp );
	/*@ End CpMem Pinned Section @*/

	/*@ Unified Memory Section @*/
	printf("\n>>> Results for Unified Memory\n");
    int *a_um, *b_um, *c_um;	

	START_CUDAEVENT
	hipMallocManaged( (void **) &a_um, size );
	hipMallocManaged( (void **) &b_um, size );
	hipMallocManaged( (void **) &c_um, size );
	STOP_AND_PRINT_CUDAEVENT([Unified Memory] memory allocation)

	START_CUDAEVENT
	for( int i = 0; i < N; i++ )
	{
		a_um[i] = b_um[i] = i;
		c_um[i] = 0;
	}
	STOP_AND_PRINT_CUDAEVENT([Unified Memory] initialize)

	START_CUDAEVENT
	/* MemCopy Part */
	STOP_AND_PRINT_CUDAEVENT([Unified Memory] MemCopy Host to Device)

	START_CUDAEVENT
	vector_add_UM<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( a_um, b_um, c_um );
	hipDeviceSynchronize();
	STOP_AND_PRINT_CUDAEVENT([Unified Memory] execution)

	START_CUDAEVENT
	/* MemCopy Part Device To Host*/
	STOP_AND_PRINT_CUDAEVENT([Unified Memory] MemCopy Device to Host)

	hipFree( a_um );
	hipFree( b_um );
	hipFree( c_um );
	/*@ End Unified Memory Section @*/	

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
} /* end main */
