
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

int main(int argc, char *argv[]) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int device;
  
  for (device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Device %d has compute capability %d.%d.\n", device,
           deviceProp.major, deviceProp.minor);
  }

  return 0;
}